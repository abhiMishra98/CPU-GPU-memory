#include <hip/hip_runtime.h>
#include <iostream>

int main(){
    int *ptr, N=20;
    int *b;
    int num = 20;
    ptr = &num;//points to a memory inside host(CPU)
    hipMalloc(&ptr, N*sizeof(int)); //points to a memory inside device(GPU)
    
    hipMemcpy(ptr, &num, sizeof(int), hipMemcpyHostToDevice);
    //without the above line, while printing *addr, value comes as 0
    //This is because, the memory region ptr inside device memory was never initialized with the value 20
    //For this, we create a memory region num inside host memory, copy it to GPU memory region ptr

    hipMallocManaged(&b, N*sizeof(int)); //unified memory

    int *addr;
    hipHostMalloc(&addr,N*sizeof(int), hipHostMallocDefault); //points to a memory inside host(CPU)
    hipPointerAttribute_t attr;

    hipPointerGetAttributes(&attr, ptr);
    std::cout<<"Address is at "<<attr.type<<std::endl; //returns 2 showing that the memory resides in GPU
    std::cout<<"Address of the variable is "<<&ptr<<std::endl;

    hipPointerGetAttributes(&attr, &(*addr));
    std::cout<<"Address is at "<<attr.type<<std::endl; //returns 1 showing that the memory resides in host (CPU)
    std::cout<<"Address of the variable is "<<&addr<<std::endl;

    hipPointerGetAttributes(&attr, &(*b));
    std::cout<<"Address is at "<<attr.type<<std::endl; //returns 3 showing that the memory resides in unified memory
    std::cout<<"Address of the variable is "<<&b<<std::endl;

    hipMemcpy(addr,ptr,N*sizeof(int),hipMemcpyDeviceToHost);
    hipPointerGetAttributes(&attr, addr);
    std::cout<<"After copying mem from device to host"<<std::endl;
    std::cout<<"Address is at "<<attr.type<<std::endl; //returns 2 showing that the memory resides in GPU
    std::cout<<"Address of the variable is "<<&addr<<std::endl;

    //std::cout<<"Value is "<<*ptr<<std::endl; 
    //Throws segmentation fault since ptr now points to a memory region inside GPU address space, de-refercing it 
    //tells the CPU to de-reference a memory region outside CPU space which actually doesn't exist
    //De-referencing of memory regions inside GPU(device) cannot be done from CPU(host).
    //The memory regions inside GPU needs to be moved to CPU to be seen.


    std::cout<<"Value is "<<*addr<<std::endl; 


    hipFree(ptr);
    hipFree(b);
    hipFree(addr);
    
    return 0;
}